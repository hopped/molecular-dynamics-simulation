#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2014 Mohamad Amer Wafai <amerwafai@gmail.com>
 * Amer Wafai
 * 07.08.2012
 */
#include "moleculecontainer/GPU/basicN2/basicN2.cuh"
__global__ void basicN2_reset_forces_and_momenta_kernel(real *fX,real *fY,real *fZ, long DN);
__global__ void basicN2_calc_forces_kernel(real *rX,real *rY,real *rZ,real *fX,real *fY, real *fZ, real *hrX,real *hrY, real *hrZ,real *U_pot, long DN, long DHN);
inline __device__ double atomicAdd(double *address, double inc );

__constant__ real Dsigma2;
__constant__ real Depsilon24;
__constant__ real Dcutoff_radius_sq;

#ifdef USE_STREAMS
hipStream_t *streams;
#endif

#define BSX 128

dim3 BlockSz(BSX, 1, 1);
dim3 GridSz(1, 1, 1);



__global__ void basicN2_calc_forces_kernel(real *rX,real *rY,real *rZ,real *fX,real *fY, real *fZ, real *hrX,real *hrY, real *hrZ,real *U_pot, long DN, long DHN){
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ real sh_buf[];
    for (; i < DN; i += blockDim.x * gridDim.x) {
        real U_pot_tmp = 0.;
        real dr[3];
        real dr2;
        real lj6;
        real lj12;
        real lj12m6;
        real u_pot;
        real factor;
        real invdr2;
        sh_buf[threadIdx.x] = rX[i];
        sh_buf[threadIdx.x + blockDim.x] = rY[i];
        sh_buf[threadIdx.x + 2 * blockDim.x] = rZ[i];
        __syncthreads();
        /* Calculate all interactions between real molecules. */
        for (int j = i+1; j < DN; j++) {
            dr[0] = rX[j] - sh_buf[threadIdx.x ];
            dr[1] = rY[j] - sh_buf[threadIdx.x + blockDim.x];
            dr[2] = rZ[j] - sh_buf[threadIdx.x + 2 * blockDim.x];

            dr2 = 0.;
            dr2 = dr[0] * dr[0] + dr[1] * dr[1] + dr[2] * dr[2];

            if (dr2 > Dcutoff_radius_sq) {
                continue;
            }

            invdr2 = 1. / dr2;

            /* Lennard Jones interaction forces */
            lj6 = Dsigma2 * invdr2;
            lj6 = lj6 * lj6 * lj6;
            lj12 = lj6 * lj6;
            lj12m6 = lj12 - lj6;
            u_pot  = Depsilon24 * lj12m6;
            factor = Depsilon24 * ( lj12 + lj12m6 ) * invdr2;

            atomicAdd(&fX[i],-1* factor * dr[0]);
            atomicAdd(&fY[i],-1* factor * dr[1]);
            atomicAdd(&fZ[i],-1* factor * dr[2]);
            atomicAdd(&fX[j], factor * dr[0]);
            atomicAdd(&fY[j], factor * dr[1]);
            atomicAdd(&fZ[j], factor * dr[2]);
            U_pot_tmp += u_pot;
        }
    
        for (int j = 0; j < DHN; j++) {
            dr[0] = hrX[j] - sh_buf[threadIdx.x];
            dr[1] = hrY[j] - sh_buf[threadIdx.x + blockDim.x];
            dr[2] = hrZ[j] - sh_buf[threadIdx.x + 2 * blockDim.x];

            dr2 = 0.;
            dr2 = dr[0] * dr[0] + dr[1] * dr[1] + dr[2] * dr[2];
            if (dr2 > Dcutoff_radius_sq) {
                continue;
            }

            invdr2 = 1. / dr2;

            /* Lennard Jones interaction forces */

            lj6 = Dsigma2 * invdr2;
            lj6 = lj6 * lj6 * lj6;
            lj12 = lj6 * lj6;
            lj12m6 = lj12 - lj6;
            u_pot  = Depsilon24 * lj12m6;
            factor = Depsilon24 * ( lj12 + lj12m6 ) * invdr2;

            atomicAdd(&fX[i],-1* factor * dr[0]);
            atomicAdd(&fY[i],-1* factor * dr[1]);
            atomicAdd(&fZ[i],-1* factor * dr[2]);
            U_pot_tmp += u_pot * .5;
        }
        U_pot_tmp/=6.0;
        atomicAdd(U_pot, U_pot_tmp);
    }
}

__global__ void basicN2_reset_forces_and_momenta_kernel(real *fX,real *fY,real *fZ, long DN){
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    for (; i < DN; i += blockDim.x * gridDim.x) {
        fX[i] = 0.;
        fY[i] = 0.;
        fZ[i] = 0.;
    }
}
inline __device__ double atomicAdd(double *address, double value)
{
    unsigned long long oldval, newval, assumed;

    oldval = __double_as_longlong(*address);
    do {
        assumed = oldval;
        newval = __double_as_longlong(__longlong_as_double(oldval) + value);
        oldval = atomicCAS((unsigned long long *)address, assumed, newval);
    } while (assumed != oldval);
    return __longlong_as_double(oldval);
}


void cuBasicN2_set_constants(long N){
    BlockSz.x = BSX;
    GridSz.x = ( N + BlockSz.x - 1 ) / BlockSz.x;
    cudaVerify(hipMemcpyToSymbolAsync(HIP_SYMBOL(Dsigma2), &sigma2, sizeof( real ),0,hipMemcpyHostToDevice));
    cudaVerify(hipMemcpyToSymbolAsync(HIP_SYMBOL(Depsilon24), &epsilon24, sizeof( real ),0, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpyToSymbolAsync(HIP_SYMBOL(Dcutoff_radius_sq), &config.cutoff_radius_sq, sizeof( real ),0, hipMemcpyHostToDevice));
}

void cuBasicN2_copyin(real *rX,real *rY,real *rZ,real *hrX,real *hrY,real *hrZ,real *drX,real *drY,real *drZ,real *dhrX,real *dhrY,real *dhrZ,long DN,long DHN, int stream_id){
#ifdef USE_STREAMS
	cudaVerify(hipMemcpyAsync(drX,rX,DN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
	cudaVerify(hipMemcpyAsync(drY,rY,DN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
	cudaVerify(hipMemcpyAsync(drZ,rZ,DN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
	cudaVerify(hipMemcpyAsync(dhrX,hrX,DHN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
	cudaVerify(hipMemcpyAsync(dhrY,hrY,DHN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
	cudaVerify(hipMemcpyAsync(dhrZ,hrZ,DHN*sizeof(real),hipMemcpyHostToDevice, streams[stream_id]));
#else
	cudaVerify(hipMemcpyAsync(drX,rX,DN*sizeof(real),hipMemcpyHostToDevice));
	cudaVerify(hipMemcpyAsync(drY,rY,DN*sizeof(real),hipMemcpyHostToDevice));
	cudaVerify(hipMemcpyAsync(drZ,rZ,DN*sizeof(real),hipMemcpyHostToDevice));
	cudaVerify(hipMemcpyAsync(dhrX,hrX,DHN*sizeof(real),hipMemcpyHostToDevice));
	cudaVerify(hipMemcpyAsync(dhrY,hrY,DHN*sizeof(real),hipMemcpyHostToDevice));
	cudaVerify(hipMemcpyAsync(dhrZ,hrZ,DHN*sizeof(real),hipMemcpyHostToDevice));
#endif
}
void cuBasicN2_reset_forces(real *f,long *count,long N, int stream_id){
#ifdef USE_STREAMS
    cudaVerifyKernel((basicN2_reset_forces_and_momenta_kernel<<<GridSz, BlockSz,0,streams[stream_id]>>>( &f[stream_id*3*N],&f[stream_id*3*N+N],&f[stream_id*3*N+N*2], count[stream_id])));
#else
    cudaVerifyKernel((basicN2_reset_forces_and_momenta_kernel<<<GridSz, BlockSz>>>( &f[stream_id*3*N], &f[stream_id*3*N+N], &f[stream_id*3*N+N*2], count[stream_id])));
#endif
}
void cuBasicN2_calc_forces(real *r,real *f,real *hr, real *U_pot, long *count,long *hcount,long N, long hN, int stream_id){
#ifdef USE_STREAMS
//    cudaVerifyKernel((basicN2_reset_forces_and_momenta_kernel<<<GridSz, BlockSz,0,streams[stream_id]>>>( &fX[stream_id*N],&fY[stream_id*N],&fZ[stream_id*N], count[stream_id])));
    cudaVerifyKernel(( basicN2_calc_forces_kernel << < GridSz, BlockSz,3*BlockSz.x*sizeof(real), streams[stream_id] >> >( &r [stream_id*3* N], &r [stream_id*3*N + N], &r [stream_id*3*N + N*2], 
															  &f [stream_id*3* N], &f [stream_id*3*N + N], &f [stream_id*3*N + N*2], 
                                                                                                                          &hr[stream_id*3*hN], &hr[stream_id*3*hN+hN], &hr[stream_id*3*hN+hN*2], 
															  &U_pot[stream_id], count[stream_id], hcount[stream_id] )));
#else
//    cudaVerifyKernel((basicN2_reset_forces_and_momenta_kernel<<<GridSz, BlockSz>>>( &fX[stream_id*N], &fY[stream_id*N], &fZ[stream_id*N], count[stream_id])));
    cudaVerifyKernel(( basicN2_calc_forces_kernel << < GridSz, BlockSz,3*BlockSz.x*sizeof(real) >> >( &r [stream_id*3*N ], &r [stream_id*3*N +N], &r [stream_id*3*N +N*2], 
												      &f [stream_id*3*N ], &f [stream_id*3*N +N], &f [stream_id*3*N +N*2], 
                                                                                                      &hr[stream_id*3*hN], &hr[stream_id*3*hN+N], &hr[stream_id*3*hN+N*2], 
												      &U_pot[stream_id], count[stream_id], hcount[stream_id] )));
#endif
}
void cuBasicN2_Sync(){
    hipDeviceSynchronize();
}
#ifdef USE_STREAMS
void cuBasicN2_create_streams(int N){
    if (( streams = (hipStream_t *)malloc(N * sizeof( hipStream_t ))) == NULL) {
        printf("Sorry there is no enough memory for stream's array");
    }
    for (int i = 0; i < N; i++) {
        hipStreamCreate(&streams[i]);
    }
}

void cuBasicN2_destroy_streams(int N){
    for (int i = 0; i < N; i++) {
        hipStreamDestroy(streams[i]);
    }
}

void HDcopyAsyncStreams(void * device, void * host, size_t size,int stream_id){
    cudaVerify(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice,streams[stream_id]));
}

void DHcopyAsyncStreams(void * host, void * device, size_t size, int stream_id){
    cudaVerify(hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost,streams[stream_id]));
}

#endif

