/*
 * Copyright (c) 2012-2014 Mohamad Amer Wafai <amerwafai@gmail.com>
 * Amer Wafai
 * 04.06.2012
 */
#include <stdio.h>
#include <stdlib.h>
#include "GPU/cuda-utils.cuh"

void Dmalloc(void ** device, size_t size){
    cudaVerify(hipMalloc(device, size));
}

void HmallocP(void ** host, size_t size){
    cudaVerify(hipHostMalloc(host, size));
}

void HDcopy(void * device, void * host, size_t size){
    cudaVerify(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
}

void DHcopy(void * host, void * device, size_t size){
    cudaVerify(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void HDcopyAsync(void * device, void * host, size_t size){
    cudaVerify(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
}

void DHcopyAsync(void * host, void * device, size_t size){
    cudaVerify(hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost));
}

void DDcopy(void * dev2, void * dev1, size_t size){
    cudaVerify(hipMemcpy(dev2, dev1, size, hipMemcpyDeviceToDevice));
}

void HHcopy(void * host2, void * host1, size_t size){
    cudaVerify(hipMemcpy(host2, host1, size, hipMemcpyHostToHost));
}

void Dfree(void * device){
    cudaVerify(hipFree(device));
    device = NULL;
}

void Hfree(void * host){
    cudaVerify(hipHostFree(host));
    host = NULL;
}

void HDSync(){
    hipDeviceSynchronize();
}
